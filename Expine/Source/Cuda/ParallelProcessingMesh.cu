#include "hip/hip_runtime.h"
#include "ParallelProcessingMesh.cuh"
#include <hip/hip_vector_types.h>

#include "Cuda.h"

__global__ void ComputeNormals(float3 * MeshNormals, float3 * MeshVertexPositions, int * Indices, int MeshFaceCount, int MeshVertexCount)
{
	for (int I = 0; I < MeshFaceCount; ++I)
	{
		int I0 = 0;
		int I1 = 1;
		int I2 = 2;

		float3 P0 = MeshVertexPositions[I0];
		float3 P1 = MeshVertexPositions[I1];
		float3 P2 = MeshVertexPositions[I2];

		float3 D0 = P0 - P1;
		float3 D1 = P2 - P1;
	
		float3 D = normalize(cross(D0, D1));

		MeshNormals[I0] += D;
		MeshNormals[I1] += D;
		MeshNormals[I2] += D;
	}

	for (int I = 0; I < MeshVertexCount; ++I)
	{
		MeshNormals[I] = normalize(MeshNormals[I]);
	}
}

void FComputeNormals(float3 * MeshNormals, float3 * MeshVertexPositions, int * Indices, int MeshFaceCount, int MeshVertexCount)
{
	ComputeNormals CudaKernelArgs2(1, 1) (MeshNormals, MeshVertexPositions, Indices, MeshFaceCount, MeshVertexCount);
}